#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// TODO: create fixed topology

// implementation should be agnostic to the node and its position

// should generalize the cases of root, 1 children, 2 children, leaf

// overlap reduction and broadcast under multiport simultaneous send-recieve model




// Vanilla C-Cube, no detours, chaining, tricks or any funny business


struct Node {
    hipStream_t R_stream; 
    hipStream_t B_stream;

    int parent;
    int left; //order [left, right].
    int right;

    int* r_lock;
    int* b_lock;

    int* r_done;
    int* b_done;

    
    float *buffer; 
}

struct Node tree[4];


// prototype for 4 node DGX-1|||| MAKES 0 FUCKING SENSE WHEN TOPOLOGY IS FULLY CONNECTED, EXACTLY 0 BENEFIT OVER 2TREE!!!!
void createCommunicator(){
    /*
    Single Tree logical topology
            0
            |
            2
           / \
          1   3
    */
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[0].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[0].B_stream), hipStreamNonBlocking);
    tree[0].left = 2;
    tree[0].right = -1;
    tree[0].parent  = -1;

    hipSetDevice(1);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[1].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[1].B_stream), hipStreamNonBlocking);
    tree[1].left = -1;
    tree[1].right = -1;
    tree[1].parent = 2;

    hipSetDevice(2);
    hipDeviceEnablePeerAccess(1,0);
    hipDeviceEnablePeerAccess(3,0);
    hipStreamCreateWithFlags(&(tree[2].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[2].B_stream), hipStreamNonBlocking);
    tree[2].left = 1;
    tree[2].right = 3;
    tree[2].parent = 0;
    
    hipSetDevice(3);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[3].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[3].B_stream), hipStreamNonBlocking);
    tree[3].left = -1;
    tree[3].right = -1;
    tree[3].parent = 2;
}



// define in-place operation for now
void allreduce(void* sendbuff, void* recvbuff, int message_size, int chunk_size){
    // multiprocess function
    // create n threads, each launching reduce_kernel and broadcast_kernel on every device
    // using tree struct
    // number of threads should be equal or close to the chunk size
}


// adapt the C-Cube under the 3 port simultaneous send-recieve model.
// two streams: reduce and broadcast
// tree should be a structure visible and referenceable from the device, not only host controlled

__global__ void reduce_kernel(float* self_buff, 
                              float* left_buff, 
                              float* right_buff, 
                              volatile int* r_lock_self, 
                              int* r_lock_parent,
                              volatile int* r_done_self,
                              int* r_done_left,
                              int* r_done_right,
                              int* b_lock_left,
                              int* b_lock_right,
                              int num_chunks)
{

    // grid size = number of elements in a chunk
    
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int gsize = gridDim.x*blockDim.x; 

    int i=0;
    int index = 0;

    //data-independent conditioning, so no branch divergence (?)
    if(parent_buff){
        // not root
        if(left_buff && right_buff){
            // two children
            for(i = 0; i<num_chunks; i++){
                index = gid + i*gsize;
                while(*r_lock_self == 0); //TODO: make each lock block dependent, for global sync
                self_buff[index] = self_buff[index] + left_buff[index] + right_buff[index];
                __syncthreads();
                if(tid == 0){
                    *r_lock_self = 0;
                    *r_lock_parent = 1;
                }   
            }
            if(tid==0){
                *r_done_left = 1;
                *r_done_right = 1;
            }
        }
        else if (left_buff){
            // one children
            for(i=0; i<num_chunks; i++){
                index = gid + i*gsize;
                while(*r_lock_self == 0);
                self_buff[index] = self_buff[index] + left_buff[index];
                __syncthreads();
                if(tid == 0){
                    *r_lock_self = 0;
                    *r_lock_parent = 1;
                }
            }
            if (tid ==0){
                *r_done_left = 1;
            }
        }
        else{
            //leaf
            if (tid==0){
                while(*r_done_self == 0){
                    *r_lock_parent = 1;
                }
            }
        }
    }
    else{
        // root
        if(left_buff && right_buff){
            // two children
            for(i = 0; i<num_chunks; i++){
                index = gid + i*gsize;
                while(*r_lock_self == 0); //TODO: make each lock block dependent, for global sync
                self_buff[index] = self_buff[index] + left_buff[index] + right_buff[index];
                __syncthreads();
                if(tid == 0){
                    *r_lock_self = 0;
                    *b_lock_left = 1;
                    *b_lock_right = 1;
                } 
            }
            if(tid==0){
                *r_done_left = 1;
                *r_done_right = 1;
                *r_done_self = 1;
            }
        }
        else if (left_buff){
            // one child
            for(i=0; i<num_chunks; i++){
                index = gid + i*gsize;
                while(*r_lock_self == 0);
                self_buff[index] = self_buff[index] + left_buff[index];
                __syncthreads();
                if(tid == 0){
                    *r_lock_self = 0;
                    *b_lock_left = 1;
                }
            }
            if (tid ==0){
                *r_done_left = 1;
                *r_done_self = 1;
            }
        }
}

__global__ void broadcast_kernel(float* self_buff,
                                 float* parent_buff,
                                 volatile int* b_lock_self,
                                 int* b_lock_left,
                                 int* b_lock_right,
                                 int* b_done_self,
                                 int* b_done_parent,
                                 int num_chunks)
{
    
    // grid size = num of elements in a chunk
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int gsize = gridDim.x*blockDim.x; 

    int i=0;
    int index = 0;

    if(parent_buff){
        if(b_lock_left && b_lock_right){
            // two children
            for (i=0; i<num_chunks; i++){
                index = gid + i*gsize;
                while(*b_lock_self == 0);
                self_buff[index] = parent_buff[index];
                __syncthreads();
                if (tid==0){
                    *b_lock_self = 0;
                    *b_lock_left = 1;
                    *b_lock_right = 1;
                }
            }
            if (tid==0){
                *b_done_parent = 1;
            }
        }
        else if (b_lock_left){
            // one child
            for (i=0; i<num_chunks; i++){
                index = gid + i*gsize;
                while(*b_lock_self == 0);
                self_buff[index] = parent_buff[index];
                __syncthreads();
                if (tid==0){
                    *b_lock_self = 0;
                    *b_lock_left = 1;
                }
            }
            if (tid==0){
                *b_done_parent = 1;
            }
        }
        else{
            // leaf
            for (i=0; i<num_chunks; i++){
                index = gid + i*gsize;
                while(*b_lock_self == 0);
                self_buff[index] = parent_buff[index];
                __syncthreads();
                if (tid==0){
                    *b_lock_self = 0;
                }
            }
            if (tid==0){
                *b_done_parent = 1;
                *b_done_self = 1;
            }
        }
    }

    // root: do nothing

}

