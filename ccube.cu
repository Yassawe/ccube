#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// TODO: create fixed topology

// implementation should be agnostic to the node and its position

// should generalize the cases of root, 1 children, 2 children, leaf

// overlap reduction and broadcast under multiport simultaneous send-recieve model




// Vanilla C-Cube, no detours, chaining, tricks or any funny business


struct Node {
    hipStream_t R_stream; 
    hipStream_t B_stream;

    int parent;
    int left; //order [left, right].
    int right;
    
    float *buffer; 
}

struct Node tree[4];


// prototype for 4 node DGX-1|||| MAKES 0 FUCKING SENSE WHEN TOPOLOGY IS FULLY CONNECTED, EXACTLY 0 BENEFIT OVER 2TREE!!!!
void createCommunicator(){
    /*
    Single Tree logical topology
            0
            |
            2
           / \
          1   3
    */
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[0].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[0].B_stream), hipStreamNonBlocking);
    tree[0].left = 2;
    tree[0].right = -1;
    tree[0].parent  = -1;

    hipSetDevice(1);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[1].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[1].B_stream), hipStreamNonBlocking);
    tree[1].left = -1;
    tree[1].right = -1;
    tree[1].parent = 2;

    hipSetDevice(2);
    hipDeviceEnablePeerAccess(1,0);
    hipDeviceEnablePeerAccess(3,0);
    hipStreamCreateWithFlags(&(tree[2].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[2].B_stream), hipStreamNonBlocking);
    tree[2].left = 1;
    tree[2].right = 3;
    tree[2].parent = 0;
    
    hipSetDevice(3);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[3].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[3].B_stream), hipStreamNonBlocking);
    tree[3].left = -1;
    tree[3].right = -1;
    tree[3].parent = 2;
}




void allreduce(void* sendbuff, void* recvbuff, int count, int chunk_size){
    // multiprocess function
    // create n threads, each launching reduce_kernel and broadcast_kernel on every device
    // using tree struct

}


// adapt the C-Cube under the 3 port simultaneous send-recieve model.
// two streams: reduce and broadcast
// tree should be a structure visible and referenceable from the device, not only host controlled

__global__ void reduce_kernel(float* self_buff, float* parent_buff, float* left_buff, float* right_buff, int chunksize){ //make it agnostic to where it runs

    

}

__global__ void broadcast_kernel(float* self_buff, float* parent_buff, float* left_buff, float* right_buff, int chunksize){


}

