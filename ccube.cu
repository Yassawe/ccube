#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// TODO: create fixed topology

// implementation should be agnostic to the node and its position

// should generalize the cases of root, 1 children, 2 children, leaf

// overlap reduction and broadcast under multiport simultaneous send-recieve model




// Vanilla C-Cube, no detours, chaining, tricks or any funny business


struct Node {
    hipStream_t R_stream; 
    hipStream_t B_stream;

    int num_c;
    int c[2]; //[left, right] order
    int p;
    
    float *buffer; // do i decide on simultaneous send-recieve model? NO!!!
}

struct Node tree[4];


// prototype for 4 node DGX-1|||| MAKES 0 FUCKING SENSE WHEN TOPOLOGY IS FULLY CONNECTED, EXACTLY 0 BENEFIT OVER 2TREE!!!!
void createCommunicator(){
    /*
    Single Tree logical topology
            0
            |
            2
           / \
          1   3
    */
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[0].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[0].B_stream), hipStreamNonBlocking);
    tree[0].num_c = 1;
    tree[0].c[0] = 2;
    tree[0].p  = -1;

    hipSetDevice(1);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[1].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[1].B_stream), hipStreamNonBlocking);
    tree[1].num_c = 0;
    tree[1].p = 2;

    hipSetDevice(2);
    hipDeviceEnablePeerAccess(1,0);
    hipDeviceEnablePeerAccess(3,0);
    hipStreamCreateWithFlags(&(tree[2].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[2].B_stream), hipStreamNonBlocking);
    tree[2].num_c = 2;
    tree[2].c[0] = 1;
    tree[2].c[1] = 3;
    
    hipSetDevice(3);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[3].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[3].B_stream), hipStreamNonBlocking);
    tree[3].num_c = 0;
    tree[3].p = 2;
}


// adapt the 2 port simultaneous send-recieve model.