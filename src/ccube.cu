#include "hip/hip_runtime.h"
#include "ccube.h"
#include <hip/hip_runtime.h>

void createCommunicator(struct Node* tree){
    /*
    simple tree
    
            0
            |
            2
           / \
          1   3
    */

    hipSetDevice(0);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[0].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[0].B_stream), hipStreamNonBlocking);
    tree[0].left = 2;
    tree[0].right = -1;
    tree[0].parent  = -1;
    allocateLocks(tree, 0);


    hipSetDevice(1);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[1].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[1].B_stream), hipStreamNonBlocking);
    tree[1].left = -1;
    tree[1].right = -1;
    tree[1].parent = 2;
    allocateLocks(tree, 1);

    hipSetDevice(2);
    hipDeviceEnablePeerAccess(0,0);
    hipDeviceEnablePeerAccess(1,0);
    hipDeviceEnablePeerAccess(3,0);
    hipStreamCreateWithFlags(&(tree[2].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[2].B_stream), hipStreamNonBlocking);
    tree[2].left = 1;
    tree[2].right = 3;
    tree[2].parent = 0;
    allocateLocks(tree, 2);

    hipSetDevice(3);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[3].R_stream), hipStreamNonBlocking);
    hipStreamCreateWithFlags(&(tree[3].B_stream), hipStreamNonBlocking);
    tree[3].left = -1;
    tree[3].right = -1;
    tree[3].parent = 2;
    allocateLocks(tree, 3);
}

void killCommunicator(struct Node* tree){
    for(int i=0; i<P; i++){
        hipSetDevice(i);
        hipFree(tree[i].r_lock);
        hipFree(tree[i].b_lock);
        hipFree(tree[i].r_ready);
        hipFree(tree[i].b_ready);
        hipStreamDestroy(tree[i].R_stream);
        hipStreamDestroy(tree[i].B_stream);
        for (int j = 0; j<P; j++){
            hipDeviceDisablePeerAccess(j);
        }
    }
}

__global__ void reduce_kernel(int parent,
                              int left,
                              int right, 
                              volatile int* r_lock_self, 
                              volatile int* r_lock_parent,
                              volatile int* r_ready,
                              volatile int* r_ready_left,
                              volatile int* r_ready_right,
                              volatile int* b_lock_left,
                              volatile int* b_lock_right,
                              volatile int* b_ready,
                              float* self_buff, 
                              float* left_buff, 
                              float* right_buff,
                              int which, // 0 if self is left, 1 if right
                              int num_chunks)
{   
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = bid*blockDim.x + tid;
    int gsize = gridDim.x*blockDim.x;
    int i=0;
    int index = 0;
    if (parent == -1){
        //root
        if (left == -1 && right == -1){
            //no children
            return;
        }
        else if(right==-1){
            //one child
            for(i=0;i<num_chunks;i++){
                index = gsize*i+gid;
                if(tid==0) r_ready_left[bid] = 1;
                while(r_lock_self[2*bid]==0);
                self_buff[index] = self_buff[index]+left_buff[index];
                __syncthreads();
                if (tid == 0) r_lock_self[2*bid] = 0;
                while(b_ready[2*bid]==0);
                if (tid == 0){
                    b_ready[2*bid] = 0;
                    b_lock_left[bid] = 1;
                }
            }
        }
        else{
            //two children
            for(i=0;i<num_chunks;i++){
                index = gsize*i+gid;
                if (tid == 0){
                    r_ready_left[bid] = 1;
                    r_ready_right[bid] = 1;
                }
                while(r_lock_self[2*bid]==0 || r_lock_self[2*bid+1]==0);
                self_buff[index] = self_buff[index]+left_buff[index]+right_buff[index];
                __syncthreads();
                if (tid == 0){
                    r_lock_self[2*bid] = 0;
                    r_lock_self[2*bid+1] = 0;
                }
                while(b_ready[2*bid]==0 || b_ready[2*bid+1]==0);
                if(tid==0){
                    b_ready[2*bid] = 0;
                    b_ready[2*bid+1] = 0;
                    b_lock_left[bid] = 1;
                    b_lock_right[bid] = 1;
                }
            }
        }
    }
    else{
        //non-root
        if (left == -1 && right == -1){
            //no children
            for(i=0; i<num_chunks; i++){
                while(r_ready[bid]==0);
                if (tid == 0){
                    r_lock_parent[2*bid+which]=1;
                    r_ready[bid] = 0;
                }
            }
        }
        else if(right==-1){
            //one child
            for(i=0; i<num_chunks; i++){
                index = gsize*i+gid;
                if (tid == 0) r_ready_left[bid] = 1;
                while(r_lock_self[2*bid]==0);
                self_buff[index] = self_buff[index]+left_buff[index];
                __syncthreads();
                if (tid == 0) r_lock_self[2*bid] = 0;
                while(r_ready[bid]==0);
                if (tid == 0){
                    r_lock_parent[2*bid+which]=1;
                    r_ready[bid] = 0;
                }
            }
        }
        else{
            //two children
            for(i=0; i<num_chunks; i++){
                index = gsize*i+gid;
                if (tid == 0){
                    r_ready_left[bid] = 1;
                    r_ready_right[bid] = 1;
                } 
                while(r_lock_self[2*bid]==0 || r_lock_self[2*bid+1]==0);
                self_buff[index] = self_buff[index]+left_buff[index]+right_buff[index];
                __syncthreads();
                if (tid == 0){
                    r_lock_self[2*bid] = 0;
                    r_lock_self[2*bid+1] = 0;
                } 
                while(r_ready[bid]==0);
                if (tid == 0){
                    r_lock_parent[2*bid+which]=1;
                    r_ready[bid] = 0;
                }
            }
        }
    }
}

__global__ void broadcast_kernel(int parent,
                                 int left,
                                 int right,
                                 volatile int* b_lock_self,
                                 volatile int* b_lock_left,
                                 volatile int* b_lock_right,
                                 volatile int* b_ready,
                                 volatile int* b_ready_parent,
                                 float* self_buff,
                                 float* parent_buff,
                                 int which,
                                 int num_chunks){

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = bid*blockDim.x + tid;
    int gsize = gridDim.x*blockDim.x;
    int i=0;
    int index = 0;
    if (parent==-1){
        //root
        return; //root does nothing
    }
    else{
        // non-root
        if(left == -1 && right == -1){
            // no children
            for(i=0; i<num_chunks; i++){
                index = gsize*i + gid;
                if (tid == 0) b_ready_parent[2*bid+which] = 1;
                while(b_lock_self[bid]==0);
                self_buff[index] = parent_buff[index];
                __syncthreads();
                if (tid == 0) b_lock_self[bid] = 0;
            }
        }
        else if (right == -1){
            // one child
            for(i=0;i<num_chunks;i++){
                index = gsize*i+gid;
                if (tid == 0) b_ready_parent[2*bid+which] = 1;
                while(b_lock_self[bid]==0);
                self_buff[index] = parent_buff[index];
                __syncthreads();
                if (tid == 0) b_lock_self[bid] = 0;
                while(b_ready[2*bid]==0);
                if(tid == 0){
                    b_ready[2*bid] = 0;
                    b_lock_left[bid] = 1;
                }
            }
        }
        else{
            // two children
            for(i=0;i<num_chunks;i++){
                index = gsize*i+gid;
                if (tid == 0) b_ready_parent[2*bid+which] = 1;
                while(b_lock_self[bid]==0);
                self_buff[index] = parent_buff[index];
                __syncthreads();
                if (tid == 0) b_lock_self[bid] = 0;
                while(b_ready[2*bid]==0 || b_ready[2*bid+1]==0);
                if(tid == 0){
                    b_ready[2*bid] = 0;
                    b_ready[2*bid+1] = 0;
                    b_lock_left[bid] = 1;
                    b_lock_right[bid] = 1;
                }
            }
        }
    }
}


int launch(struct Node* tree, int rank, int num_chunks){
    hipSetDevice(rank);
    int parent = tree[rank].parent;
    int left = tree[rank].left;
    int right = tree[rank].right;
    int which = 0;

    if (parent != -1){
        which = (rank == tree[parent].right) ? 1 : 0;
    }
 
    reduce_kernel<<<NUM_BLOCKS, BLOCK_SIZE, 0, tree[rank].R_stream>>>(parent,
                                                                    left,
                                                                    right, 
                                                                    tree[rank].r_lock,
                                                                    (parent!=-1) ? tree[parent].r_lock : NULL,
                                                                    tree[rank].r_ready,
                                                                    (left!=-1) ? tree[left].r_ready : NULL,
                                                                    (right!=-1) ? tree[right].r_ready : NULL,
                                                                    (left!=-1) ? tree[left].b_lock : NULL,
                                                                    (right!=-1) ? tree[right].b_lock : NULL,
                                                                    tree[rank].b_ready,
                                                                    tree[rank].buffer,
                                                                    (left!=-1) ? tree[left].buffer : NULL,
                                                                    (right!=-1) ? tree[right].buffer : NULL,
                                                                    which,
                                                                    num_chunks);

    broadcast_kernel<<<NUM_BLOCKS, BLOCK_SIZE, 0, tree[rank].B_stream>>>(parent,
                                                                        left,
                                                                        right,
                                                                        tree[rank].b_lock,
                                                                        (left!=-1) ? tree[left].b_lock : NULL,
                                                                        (right!=-1) ? tree[right].b_lock : NULL,
                                                                        tree[rank].b_ready,
                                                                        (parent!=-1) ? tree[parent].b_ready : NULL,
                                                                        tree[rank].buffer,
                                                                        (parent!=-1) ? tree[parent].buffer : NULL,
                                                                        which,
                                                                        num_chunks);
    
    hipDeviceSynchronize();
    return 0;
}
