#include "hip/hip_runtime.h"
#include "ccube.h"
#include <hip/hip_runtime.h>

#define CUDAERRORCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void allocate_lock(int* pointer){
    hipMalloc((void **)&pointer, sizeof(int));
    hipMemset(pointer, 0, sizeof(int));
}

void createCommunicator(struct Node* tree){
    /*
    simple pipeline for debugging.
    single block for debugging.
    
            0
            |
            1
            |
            2
    */

    hipSetDevice(0);
    hipDeviceEnablePeerAccess(1,0);
    hipStreamCreateWithFlags(&(tree[0].stream), hipStreamNonBlocking);
    tree[0].child = 1;
    tree[0].parent  = -1;
    allocate_lock(tree[0].lock);
    allocate_lock(tree[0].ready);


    hipSetDevice(1);
    hipDeviceEnablePeerAccess(2,0);
    hipDeviceEnablePeerAccess(0,0);
    hipStreamCreateWithFlags(&(tree[1].stream), hipStreamNonBlocking);
    tree[1].child = 2;
    tree[1].parent = 0;
    allocate_lock(tree[1].lock);
    allocate_lock(tree[1].ready);

    hipSetDevice(2);
    hipDeviceEnablePeerAccess(1,0);
    hipStreamCreateWithFlags(&(tree[2].stream), hipStreamNonBlocking);
    tree[2].child = -1;
    tree[2].parent = 1;
    allocate_lock(tree[2].lock);
    allocate_lock(tree[2].ready);

}

void killCommunicator(struct Node* tree){
    for(int i=0; i<P; i++){
        hipSetDevice(i);
        hipFree(tree[i].lock);
        hipFree(tree[i].ready);
        hipStreamDestroy(tree[i].stream);
        for (int j = 0; j<P; j++){
            hipDeviceDisablePeerAccess(j);
        }
    }
}



int launch(struct Node* tree, int rank, int parent, int left, int right, int num_chunks){

    hipSetDevice(rank);

 
    reduce_kernel<<<(CHUNK_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE, 0, tree[rank].R_stream>>>(parent,
                                                                                                left,
                                                                                                right,
                                                                                                tree[rank].buffer,
                                                                                                (left == -1) ? NULL : tree[left].buffer,
                                                                                                (right == -1) ? NULL : tree[right].buffer,
                                                                                                tree[rank].r_lock,
                                                                                                (parent == -1) ? NULL : tree[parent].r_lock,
                                                                                                tree[rank].r_done,
                                                                                                (left == -1) ? NULL : tree[left].r_done,
                                                                                                (right == -1) ? NULL : tree[right].r_done,
                                                                                                (left == -1) ? NULL : tree[left].b_lock,
                                                                                                (right == -1) ? NULL : tree[right].b_lock,
                                                                                                num_chunks);

    CUDAERRORCHECK(hipDeviceSynchronize());

    // broadcast_kernel<<<(CHUNK_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE, 0, tree[rank].B_stream>>>(parent,
    //                                                                                                 left,
    //                                                                                                 right,
    //                                                                                                 tree[rank].buffer,
    //                                                                                                 (parent == -1) ? NULL : tree[parent].buffer,
    //                                                                                                 tree[rank].b_lock,
    //                                                                                                 (left == -1) ? NULL : tree[left].b_lock,
    //                                                                                                 (right == -1) ? NULL : tree[right].b_lock,
    //                                                                                                 tree[rank].b_done,
    //                                                                                                 (parent == -1) ? NULL : tree[parent].b_done,
    //                                                                                                 num_chunks);
    return 0;
}


//[DEBUG]

__global__ void p2p_sum(float* a, float* b, int num_chunks){
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    int gsize = gridDim.x*blockDim.x;

    int i = 0;
    int index = 0;

    for (i=0; i<num_chunks; i++){
        index = gid + i*gsize;
        a[index] = a[index] + b[index];
        __syncthreads();
    }
}

void testp2p(struct Node* tree,int rank, int peer, int num_chunks){
    hipSetDevice(rank);
    p2p_sum<<<CHUNK_SIZE/BLOCK_SIZE, BLOCK_SIZE>>>(tree[rank].buffer, tree[peer].buffer, num_chunks);
    CUDAERRORCHECK(hipDeviceSynchronize());
}