#include "hip/hip_runtime.h"
#include "ccube.h"
#include <hip/hip_runtime.h>

#define CUDAERRORCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void createCommunicator(struct Node* tree){
    /*
    simple pipeline for debugging.
    single block for debugging.
    
            0
            |
            1
            |
            2
            |
            3
    */

    hipSetDevice(0);
    hipDeviceEnablePeerAccess(1,0);
    hipStreamCreateWithFlags(&(tree[0].stream), hipStreamNonBlocking);
    tree[0].child = 1;
    tree[0].parent  = -1;
    allocateLocks(tree, 0);


    hipSetDevice(1);
    hipDeviceEnablePeerAccess(2,0);
    hipDeviceEnablePeerAccess(0,0);
    hipStreamCreateWithFlags(&(tree[1].stream), hipStreamNonBlocking);
    tree[1].child = 2;
    tree[1].parent = 0;
    allocateLocks(tree, 1);

    hipSetDevice(2);
    hipDeviceEnablePeerAccess(1,0);
    hipDeviceEnablePeerAccess(3,0);
    hipStreamCreateWithFlags(&(tree[2].stream), hipStreamNonBlocking);
    tree[2].child = 3;
    tree[2].parent = 1;
    allocateLocks(tree, 2);

    hipSetDevice(3);
    hipDeviceEnablePeerAccess(2,0);
    hipStreamCreateWithFlags(&(tree[3].stream), hipStreamNonBlocking);
    tree[3].child = -1;
    tree[3].parent = 2;
    allocateLocks(tree, 3);
}

void killCommunicator(struct Node* tree){
    for(int i=0; i<P; i++){
        hipSetDevice(i);
        hipFree(tree[i].lock);
        hipFree(tree[i].ready);
        hipStreamDestroy(tree[i].stream);
        for (int j = 0; j<P; j++){
            hipDeviceDisablePeerAccess(j);
        }
    }
}

__global__ void simple_reduce(int parent,
                              int child,
                              volatile int* lock,
                              volatile int* p_lock,
                              volatile int* ready,
                              volatile int* c_ready,
                              float* self_buffer,
                              float* child_buffer,
                              int num_chunks){
    
    
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    int gid = bid*blockDim.x + tid;
    int gsize = gridDim.x*blockDim.x;
    
    int i = 0; 
    int index = 0;
    
    if (parent ==-1){
        //root
        for(i=0; i<num_chunks; i++){
            index = gsize*i + gid;
            if(tid == 0) c_ready[bid] = 1;
                
            while(lock[bid]==0);
    
            self_buffer[index] = self_buffer[index] + child_buffer[index];
            __syncthreads();
                
            if(tid == 0) lock[bid] = 0;
        }

    }
    else{
        //non-root
        if (child ==-1){
            //leaf
            for(i=0;i<num_chunks; i++){
                while(ready[bid]==0);
                if (tid==0){
                    p_lock[bid] = 1;
                    ready[bid] = 0;
                }
            }
        }
        else{
            //non-leaf
            for(i = 0; i<num_chunks; i++){
                index = gsize*i + gid;
                if(tid == 0) c_ready[bid] = 1;
                
                while(lock[bid]==0);
    
                self_buffer[index] = self_buffer[index] + child_buffer[index];
                __syncthreads();
                
                if(tid == 0) lock[bid] = 0;
                
                while(ready[bid]==0);
    
                if(tid == 0){
                    p_lock[bid] = 1;
                    ready[bid] = 0;
                } 
            }
        }    
        
    }
    
}

int launch(struct Node* tree, int rank, int num_chunks){
    hipSetDevice(rank);

    int parent = tree[rank].parent;
    int child = tree[rank].child;

    simple_reduce<<<CHUNK_SIZE/BLOCK_SIZE, BLOCK_SIZE, 0, tree[rank].stream>>>(parent,
                                                                               child,
                                                                               tree[rank].lock,
                                                                               (parent==-1) ? NULL : tree[parent].lock,
                                                                               tree[rank].ready,
                                                                               (child==-1) ? NULL : tree[child].ready,
                                                                               tree[rank].buffer,
                                                                               (child==-1) ? NULL : tree[child].buffer,
                                                                               num_chunks);


    CUDAERRORCHECK(hipDeviceSynchronize());
    return 0;
}
