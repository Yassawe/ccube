#include "ccube.h"
#include <hip/hip_runtime.h>


// for now, allocate all ones
void allocateMemoryBuffers(struct Node* tree, int message_size){
    for(int i = 0; i<P; i++){
        hipMalloc((void **)&tree[i].buffer, message_size*sizeof(float));
        hipMemset(tree[i].buffer, 1, message_size*sizeof(float));
    }
}


void freeMemoryBuffers(struct Node* tree){
    for(int i = 0; i<P; i++){
        hipFree(tree[i].buffer);
    }
}